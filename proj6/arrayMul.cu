#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block (16, 32, 64)
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size(16K, 32K, 64K, 128K, 256K, and 512K)
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN = 0.;
const float XCMAX = 2.0;
const float YCMIN = 0.0;
const float YCMAX = 2.0;
const float RMIN = 0.5;
const float RMAX = 2.0;

__host__ void
TimeOfDaySeed()
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time(&timer);
	double seconds = difftime(timer, mktime(&y2k));
	unsigned int seed = (unsigned int)(1000.*seconds);    // milliseconds
	srand(seed);
}

__host__ float
Ranf(float low, float high)
{
	float r = (float)rand();				// 0 - RAND_MAX
	float t = r / (float)RAND_MAX;			// 0. - 1.

	return   low + t * (high - low);
}

// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void Monte_Carlo( float *A, float *B, float *R, float *C )
{
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	/*prods[tnum] = A[gid] * B[gid] * R[gid];*/

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2.*(A[gid] + B[gid]);
	float c = A[gid] * A[gid] + B[gid] * B[gid] - R[gid] * R[gid];
	float d = b*b - 4.*a*c;

	// If d is less than 0., then the circle was completely missed. (Case A) no hit.
	if (d < 0) prods[tnum] = 0;
	else {
		// hits the circle:
		// get the first intersection:
		d = sqrt(d);
		float t1 = (-b + d) / (2.*a);		// time to intersect the circle
		float t2 = (-b - d) / (2.*a);		// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

		// If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B) no hit.
		if (tmin < 0) prods[tnum] = 0;
		else {
			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - A[gid];
			float ny = ycir - B[gid];
			float n = sqrt(nx*nx + ny*ny);
			nx /= n;	// unit vector
			ny /= n;	// unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrt(inx*inx + iny*iny);
			inx /= in;	// unit vector
			iny /= in;	// unit vector

			// get the outgoing (bounced) vector:
			float dot = inx*nx + iny*ny;
			//float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
			float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

											// find out if it hits the infinite plate:
			float t = (0. - ycir) / outy;

			// If t is less than 0., then the reflected beam went up instead of down. no hit.
			if (t < 0) prods[tnum] = 0;

			// Otherwise, this beam hit the infinite plate. (Case D) Increment the number of hits.
			else prods[tnum] = 1;
		}
	}
	

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = prods[0];
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	/**************************Setting Up the Memory for the Arrays*************************/
	// allocate host memory:

	float * hA = new float [ SIZE ];				// xcs
	float * hB = new float [ SIZE ];				// ycs
	float *hR = new float[SIZE];					// rs
	float * hC = new float [ SIZE/BLOCKSIZE ];

	TimeOfDaySeed();		// seed the random number generator

	for( int i = 0; i < SIZE; i++ )
	{
		/*hA[i] = hB[i] = (float) sqrt(  (float)(i+1)  );*/
		hA[i] = Ranf(XCMIN, XCMAX);
		hB[i] = Ranf(YCMIN, YCMAX);
		hR[i] = Ranf(RMIN, RMAX);
	}

	// allocate device memory:

	float *dA, *dB, *dR, *dC;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsR(SIZE, 1, 1);
	dim3 dimsC( SIZE/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc(reinterpret_cast<void **>(&dR), SIZE * sizeof(float));
		checkCudaErrors(status);
	status = hipMalloc( reinterpret_cast<void **>(&dC), (SIZE/BLOCKSIZE)*sizeof(float) );
		checkCudaErrors( status );


	/*************Copying the Arrays from the Host to the Device***********/
	// copy host memory to the device:

	status = hipMemcpy( dA, hA, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, hB, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy(dR, hR, SIZE * sizeof(float), hipMemcpyHostToDevice);
		checkCudaErrors(status);


	/**********************Getting Ready to Execute***********************/
	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );


	/*****************************Executing the Kernel******************************/
	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
		Monte_Carlo <<< grid, threads >>>( dA, dB, dR, dC );
	}


	/****************************Getting the Stop Time*******************************/
	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );


	/******************************Printing the Performance*****************************/
	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double gigaMultsPerSecond = multsPerSecond / 1000000000.;
	fprintf( stderr, "\t\t\t\t\tgigaTrialsPerSecond/Second = %10.2lf\t\tArray Size = %10d\tBLOCKSIZE = %d\n", SIZE, BLOCKSIZE, gigaMultsPerSecond );


	/***************************Copying the Array from the Device to the Host*****************************/
	// copy result from the device to the host:

	status = hipMemcpy( hC, dC, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum :

	double sum = 0.;
	for(int i = 0; i < SIZE/BLOCKSIZE; i++ )
	{
		//fprintf(stderr, "hC[%6d] = %10.2f\n", i, hC[i]);
		sum += (double)hC[i];
	}
	fprintf( stderr, "sum = %10.2lf, probability = %10.2lf\n", sum, sum / double(SIZE) );


	/************************Cleaning Up******************************/
	// clean up memory:
	delete [ ] hA;
	delete [ ] hB;
	delete[] hR;
	delete [ ] hC;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree(dR);
		checkCudaErrors(status);
	status = hipFree( dC );
		checkCudaErrors( status );


	return 0;
}